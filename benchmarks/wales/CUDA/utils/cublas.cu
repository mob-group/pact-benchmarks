/* This work is a modification of code written by Jens Wetzl and Oliver Taubamann in 2012. 
 * The original work can be found here: https://github.com/jwetzl/CudaLBFGS (license: http://creativecommons.org/licenses/by/3.0/) 
 * This work is not endorsed by the authors. */

/**
 *   ___ _   _ ___   _     _       ___ ___ ___ ___
 *  / __| | | |   \ /_\   | |  ___| _ ) __/ __/ __|
 * | (__| |_| | |) / _ \  | |_|___| _ \ _| (_ \__ \
 *  \___|\___/|___/_/ \_\ |____|  |___/_| \___|___/
 *
 * File cublas.cu: Implementation of class Cublas.
 *
 **/

#include "hipblas.h"

Cublas::Cublas()
{
	CublasSafeCall( hipblasCreate(&m_cublasHandle) );
}

Cublas::~Cublas()
{
	CublasSafeCall( hipblasDestroy(m_cublasHandle) );
}

// Vector operations

void Cublas::dispatchAxpy(const size_t n, double *d_dst, const double *d_y, const double *d_x, const double *a, bool isDevicePointer) const
{
	const hipblasPointerMode_t mode = isDevicePointer ? HIPBLAS_POINTER_MODE_DEVICE
		: HIPBLAS_POINTER_MODE_HOST;

	CublasSafeCall( hipblasSetPointerMode(m_cublasHandle, mode) );

	if (d_dst != d_y)
		CudaSafeCall( hipMemcpy(d_dst, d_y, n * sizeof(double), hipMemcpyDeviceToDevice) );

	CublasSafeCall( hipblasDaxpy(m_cublasHandle, int(n), a, d_x, 1, d_dst, 1) );
}

void Cublas::dispatchScale(const size_t n, double *d_dst, const double *d_x, const double *a, bool isDevicePointer) const
{
	const hipblasPointerMode_t mode = isDevicePointer ? HIPBLAS_POINTER_MODE_DEVICE
		: HIPBLAS_POINTER_MODE_HOST;

	CublasSafeCall( hipblasSetPointerMode(m_cublasHandle, mode) );

	if (d_dst != d_x)
		CudaSafeCall( hipMemcpy(d_dst, d_x, n * sizeof(double), hipMemcpyDeviceToDevice) );

	CublasSafeCall( hipblasDscal(m_cublasHandle, int(n), a, d_dst, 1) );
}

void Cublas::dispatchDot(const size_t n, double *dst, const double *d_x, const double *d_y, bool isDstDevicePointer) const
{
	const hipblasPointerMode_t mode = isDstDevicePointer ? HIPBLAS_POINTER_MODE_DEVICE
		: HIPBLAS_POINTER_MODE_HOST;

	CublasSafeCall( hipblasSetPointerMode(m_cublasHandle, mode) );

	CublasSafeCall( hipblasDdot(m_cublasHandle, int(n), d_x, 1, d_y, 1, dst) );
}

void Cublas::dispatchNrm2(const size_t n, double *dst, const double *d_x, bool isDevicePointer) const
{
	const hipblasPointerMode_t mode = isDevicePointer ? HIPBLAS_POINTER_MODE_DEVICE
		: HIPBLAS_POINTER_MODE_HOST;

	CublasSafeCall( hipblasSetPointerMode(m_cublasHandle, mode) );

	CublasSafeCall( hipblasDnrm2(m_cublasHandle, int(n), d_x, 1, dst) );
}

